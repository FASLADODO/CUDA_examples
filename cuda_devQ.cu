// Device qurey
#include <stdio.h>

int main()
{

	int nDevices;

	hipGetDeviceCount(&nDevices);

	printf("Total number of devices %d\n", nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n",
			2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
		printf("  Number of multiprocessors:%d\n",prop.multiProcessorCount);
		printf("  Device major: %d\n", prop.major);
		printf("  Total number of cores: %d\n\n",prop.mPultiProcessorCount*128);
		printf("  Total global memory: %zu\n", prop.totalGlobalMem);
		printf("  Total amount of shared memory per block: %zu\n",prop.sharedMemPerBlock);
		printf("  Total registers per block: %d\n", prop.regsPerBlock);
		printf("  Warp size: %d\n", prop.warpSize);
		printf("  Maximum memory pitch: %u\n", prop.memPitch);
		printf("  Total amount of constant memory: %u\n", prop.totalConstMem);
	}
	return 0;
}