#include "hip/hip_runtime.h"
// (for clarity, hardcoding 128 threads/elements and omitting out-of-bounds checks)
__global__ void use_shared_memory_GPU(float *array)
{
	// local variable, private to each thread
	int i, index = threadIdx.x ;
	float average, sum = 0.0f;


	// __shared variables are visible to all threads in the thread block
	// and have the same lifetime as the thread block
	__shared__ float sh_arr[128];  

	// copy data from "array" in global memory to sh_arr in shared memory.
	// here, each thread is responsible for copying a single element.
	sh_arr[index] = array[index];

	__syncthreads();   // ensure all the writes to shared memory have completed

	// now, sh_arr is fully populated. Let's find average of all the previous elements
	for (i=0; i<index; i++) { sum += sh_arr[i];}
	average = sum / (index + 1.0f) ;

	// if array[index] is greater than the average of array[0..index-1], replace with average,
	if(array[index]>average){array[index]=average;}

	// Following code has NO EFFECT: it modifies shared memory, but
	// the resulting modified data is never copied back to global memory
	// and vanishes when the thread block completes
	sh_arr[index] = 3.14;

	__syncthreads();
}

int main(int argc,char * argv)
{
	float h_arr[128];  // convention: h_ variables live on host
	float *d_arr;      // convention: d_ variables live on the device (GPU global mem)
	
	// allocate global memory on the device, place result in "d_arr"
	hipMalloc((void **) &d_arr, sizeof(float) * 128);

	// now copy data from host memory "h_arr" to device memory "d_arr"
	cudaMemCopy((void *)d_arr, (void *)h_arr, sizeof(float)*128, hipMemcpyHostToDevice);
	
	// launch the kernel (1 block of 128 threads)
	use_shared_memory_GPU<<<1,128>>>(d_arr);

	// copy the modified array back to the host, overwriting contents of h_arr
	cudaMemCopy((void *)h_arr, (void *)d_arr, sizeof(float)*128, hipMemcpyDeviceToHost);
	
	return 0;
}
