#include "hip/hip_runtime.h"
#include <stdio.h>

#define BLOCK_WIDTH 1000
#define TOTAL_THREADS 1000000
#define BINS 4


__global__ void naive_histo(int *d_bins,const int *d_in, const int BINS_COUNT)
{
	int myId = threadIdx.x + blockDim.x * blockIdx.x ;
	int myItem = d_in[myId];
	int myBin = myItem % BINS_COUNT;
	d_bins[myBin]++;
}
__global__ void simple_histo(int *d_bins,const int *d_in, const int BINS_COUNT)
{
	int myId = threadIdx.x + blockDim.x * blockIdx.x ;
	int myItem = d_in[myId];
	int myBin = myItem % BINS_COUNT;
	atomicAdd(&d_bins[myBin],1);
}

int main(int argc, char **argv)
{
	// Array size is can be considered as number of measurements taken
	const int ARRAY_SIZE = 65536;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);
	const int BIN_COUNT = 16;
	const int BIN_BYTES = BIN_COUNT * sizeof(int);
	
	// Generate the nput array on host (some random 655536 numbers)
        int h_histo[ARRAY_SIZE];
	for (int i = 0; i< ARRAY_SIZE ; i++) {
		h_histo[i] = bit_reverse(i, log2(ARRAY_SIZE));
	}
	int h_bins[BIN_COUNT];
	for (int i = 0; i< BIN_COUNT ; i++) {
		h_bins[i] = 0;
	}

	int d_bins*;
	hipMalloc((void**)&d_bins, BIN_BYTES);
	hipMemcpy(&d_bins, &h_bins, BIN_BYTES, hipMemcpyHostToDevice);

	int d_in*;
	hipMalloc((void**)&d_in, ARRAY_BYTES);
	hipMemcpy(&d_in, &h_histo, ARRAY_BYTES, hipMemcpyHostToDevice);



	// Kernel call
	naive_histo<<<1,16>>>(&d_bins,&d_in,BIN_COUNT);

	// Copy back data to host



	// Freeup gpu memory Before exit
	hipFree(d_bins);
	hipFree(d_in);

	return 0;
}
