#include "hip/hip_runtime.h"

#include <stdio.h>

using namespace std;

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 1

__global__ void hello()
{
	printf("hello worlds! I'm thread in block %d\n", blockIdx.x);
}

int main(int argc, char **argv) {
	// Launching kernel
	hello << <NUM_BLOCKS, BLOCK_WIDTH >> > ();


	// Forceprint to flush
	hipDeviceSynchronize();
	printf("THats's all!\n");

	return 0;
}
