
#include <hip/hip_runtime.h>
#include <stdio.h>


#define NUM_BLOCKS 16
#define BLOCK_WIDTH 1


__global__ void hello()
{
	printf("I'm thread in block %d\n",blockIdx.x);
}

int main()
{

	// Launch kernel
	hello<<<NUM_BLOCKS,BLOCK_WIDTH>>>();

	// Force the printf()s to flush
	hipDeviceSynchronize();

	printf("That's all ;) ");

	return 0;
}