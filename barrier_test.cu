#include "hip/hip_runtime.h"

#include <stdio.h>

using namespace std;

#define NUM_BLOCKS 1
#define BLOCK_WIDTH 128

__global__ void hello()
{
	printf("hello worlds! I'm thread %d in block %d\n", threadIdx.x, blockIdx.x);
	int idx = threadIdx.x ;
	__shared__ int array[128];
	array[idx] = threadIdx.x ;

	printf("Before::: Array val  : %d fot tid : %d\n", array[idx],threadIdx.x);
	__syncthreads();

	if(idx < 127) {
		int temp = array[idx+1] ;
	__syncthreads();
		array[idx] =temp;
	__syncthreads();
	}
	printf("After::: Array val  : %d fot tid : %d\n", array[idx],threadIdx.x);
}

int main(int argc, char **argv) {
	// Launching kernel
	hello << <NUM_BLOCKS, BLOCK_WIDTH >> > ();


	// Forceprint to flush
	hipDeviceSynchronize();
	printf("THats's all!\n");

	return 0;
}
