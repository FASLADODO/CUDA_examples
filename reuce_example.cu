#include "hip/hip_runtime.h"
// d_out and d_in is in global memory
__global__ void global_reduce_kernel(float * d_out, float * d_in)
{
	int myId = threadIdx.x + blockDim.x * blockIdx.x ;
	int tid  = threadIdx.x ;

	// do reduction in global memory
	// Say we have 1024*1024 data chunk in global memory. i.e. d_in
	// 1024 blocks with 1024 thread will reduce to d_out of 1024 result

	// each block is responsible for 1024 chunk of floats
	for(unsigned int s = blockDim.x/2; s>0 ){

	}
}