#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gputimer.h"

#define NUM_THREADS 1000000
#define ARRAY_SIZE 10

#define BLOCK_WIDTH 1000

void print_array(int *array, int size) {
	for (int i = 0 ; i < size ; i++)
		printf("a[%d] = %d\n",i,array[i]);
}

__global__ void increment_naive(int *g) {
	// Which thread is this?
	int i = blockIdx.x * blockDim.x + threadIdx.x ;

	// each thread to increment consecutive element, wrapping at ARRAY_SIZE
	i = i % ARRAY_SIZE ;
	g[i] = g[i] + 1;
}


int main(int argc, char **argv) {
	GpuTimer timer;
	printf("%d total threads in %d blocks writing into %d array elements\n",
			NUM_THREADS, NUM_THREADS/BLOCK_WIDTH, ARRAY_SIZE);

	// declare and allocate host memmory
	int h_array[ARRAY_SIZE];
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);

	// declare, allocate and zero out GPU memory
	int *d_array;
	hipMalloc((void**) &d_array, ARRAY_BYTES);
	hipMemset((void*) d_array, 0, ARRAY_BYTES);


	// Benchmarking
	timer.Start();
	increment_naive<<<NUM_THREADS/BLOCK_WIDTH, BLOCK_WIDTH>>>(d_array);
	timer.Stop();

	// Copy back the array
	hipMemcpy(h_array,d_array,ARRAY_BYTES,hipMemcpyDeviceToHost);
 	print_array(h_array, ARRAY_SIZE);
	printf("Time lapsed = %g ms\n",timer.Elapsed());

	hipFree(d_array);

	return 0;


}
